// @file datacu.cu
// @brief Basic data structures (CUDA support)
// @author Andrea Vedaldi

/*
Copyright (C) 2015-16 Andrea Vedaldi.
All rights reserved.

This file is part of the VLFeat library and is made available under
the terms of the BSD license (see the COPYING file).
*/

#ifndef ENABLE_GPU
#error "datacu.cu cannot be compiled without GPU support"
#endif

#include "datacu.hpp"
#include "impl/blashelper.hpp"
#include <hipblas.h>

#ifndef NDEBUG
#include <iostream>
#endif

using namespace vl ;

/* -------------------------------------------------------------------
 * CudaHelper
 * ---------------------------------------------------------------- */

vl::CudaHelper::CudaHelper()
: isCublasInitialized(false), cudaStream(0)
#if ENABLE_CUDNN
, isCudnnInitialized(false), cudnnEnabled(true)
#endif
{
#if ENABLE_CUDNN
  resetCudnnConvolutionSettings() ;
#endif
}

vl::CudaHelper::~CudaHelper()
{
  clear() ;
}

void
vl::CudaHelper::clear()
{
  setStream(NULL) ;
  clearCublas() ;
#ifdef ENABLE_CUDNN
  clearCudnn() ;
#endif
}

void
vl::CudaHelper::invalidateGpu()
{
  cudaStream = NULL ;
#ifndef NDEBUG
  std::cout<<"CudaHelper::invalidateGpu()"<<std::endl ;
#endif
  isCublasInitialized = false ;
#ifdef ENABLE_CUDNN
  isCudnnInitialized = false ;
#endif
}

vl::ErrorCode
CudaHelper::setStream(hipStream_t streamId)
{
  if (isCublasInitialized) {
    hipblasStatus_t status = hipblasSetStream(cublasHandle, streamId) ;
    if (status != HIPBLAS_STATUS_SUCCESS) {
      return catchCublasError(status, __func__) ;
    }
  }
#ifdef ENABLE_CUDNN
  if (isCudnnInitialized) {
    hipdnnStatus_t status = hipdnnSetStream(cudnnHandle, streamId) ;
    if (status != HIPDNN_STATUS_SUCCESS) {
      if (isCublasInitialized) {
        // restore cuBLAS state
        hipblasSetStream(cublasHandle, this->cudaStream) ;
      }
      return catchCudnnError(status, __func__) ;
    }
  }
#endif
  this->cudaStream = streamId ;
  return VLE_Success ;
}

hipStream_t
CudaHelper::getStream() const
{
  return this->cudaStream ;
}

/* -------------------------------------------------------------------
 * getCublasHandle
 * ---------------------------------------------------------------- */

hipblasStatus_t
vl::CudaHelper::getCublasHandle(hipblasHandle_t* handle)
{
  if (!isCublasInitialized) {
    clearCublas() ;
    hipblasStatus_t status = hipblasCreate(&cublasHandle) ;
    if (status != HIPBLAS_STATUS_SUCCESS) { return status ; }

    status = hipblasSetStream(cublasHandle, cudaStream) ;
    if (status != HIPBLAS_STATUS_SUCCESS) {
      hipblasDestroy(cublasHandle) ;
      return status ;
    }
    isCublasInitialized = true ;
  }
  *handle = cublasHandle ;
  return HIPBLAS_STATUS_SUCCESS ;
}

void
vl::CudaHelper::clearCublas()
{
  if (!isCublasInitialized) { return ; }
  hipblasDestroy(cublasHandle) ;
  isCublasInitialized = false ;
}

/* -------------------------------------------------------------------
 * getCudnnHandle
 * ---------------------------------------------------------------- */

#if ENABLE_CUDNN
hipdnnStatus_t
vl::CudaHelper::getCudnnHandle(hipdnnHandle_t* handle)
{
  if (!isCudnnInitialized) {
    clearCudnn() ;
    hipdnnStatus_t stat = hipdnnCreate(&cudnnHandle) ;
    if (stat != HIPDNN_STATUS_SUCCESS) { return stat ; }
    isCudnnInitialized = true ;
  }
  *handle = cudnnHandle ;
  return HIPDNN_STATUS_SUCCESS ;
}

void
vl::CudaHelper::clearCudnn()
{
  if (!isCudnnInitialized) { return ; }
  hipdnnDestroy(cudnnHandle) ;
  isCudnnInitialized = false ;
}

bool
vl::CudaHelper::getCudnnEnabled() const
{
  return cudnnEnabled ;
}

void
vl::CudaHelper::setCudnnEnabled(bool active)
{
  cudnnEnabled = active ;
}

/* -------------------------------------------------------------------
 * cuDNN parameters
 * ---------------------------------------------------------------- */

void
vl::CudaHelper::resetCudnnConvolutionSettings()
{
  cudnnConvolutionFwdSpecificAlgo = false ;
  cudnnConvolutionFwdPreference = HIPDNN_CONVOLUTION_FWD_SPECIFY_WORKSPACE_LIMIT ;
  cudnnConvolutionFwdAlgo = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM ;
  cudnnConvolutionFwdWorkSpaceLimit = 512 * 1024 * 1024 ; // 512MB
  cudnnConvolutionFwdWorkSpaceUsed = 0 ;

  cudnnConvolutionBwdFilterSpecificAlgo = false ;
  cudnnConvolutionBwdFilterPreference = HIPDNN_CONVOLUTION_BWD_FILTER_SPECIFY_WORKSPACE_LIMIT ;
  cudnnConvolutionBwdFilterAlgo = HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_0 ;
  cudnnConvolutionBwdFilterWorkSpaceLimit = 512 * 1024 * 1024 ; // 512MB
  cudnnConvolutionBwdFilterWorkSpaceUsed = 0 ;

  cudnnConvolutionBwdDataSpecificAlgo = false ;
  cudnnConvolutionBwdDataPreference = HIPDNN_CONVOLUTION_BWD_DATA_SPECIFY_WORKSPACE_LIMIT ;
  cudnnConvolutionBwdDataAlgo = HIPDNN_CONVOLUTION_BWD_DATA_ALGO_0 ;
  cudnnConvolutionBwdDataWorkSpaceLimit = 512 * 1024 * 1024 ; // 512MB
  cudnnConvolutionBwdDataWorkSpaceUsed = 0 ;
}

void
vl::CudaHelper::setCudnnConvolutionFwdPreference(hipdnnConvolutionFwdPreference_t x,
                                                 size_t workSpaceLimit)
{
  cudnnConvolutionFwdSpecificAlgo = false ;
  cudnnConvolutionFwdPreference = x ;
  cudnnConvolutionFwdWorkSpaceLimit = workSpaceLimit ;
}

void
vl::CudaHelper::setCudnnConvolutionFwdAlgo(hipdnnConvolutionFwdAlgo_t x)
{
  cudnnConvolutionFwdSpecificAlgo = true ;
  cudnnConvolutionFwdAlgo = x ;
}

size_t
vl::CudaHelper::getCudnnConvolutionFwdWorkSpaceUsed() const
{
  return cudnnConvolutionFwdWorkSpaceUsed ;
}

void
vl::CudaHelper::setCudnnConvolutionBwdFilterPreference(hipdnnConvolutionBwdFilterPreference_t x,
                                                       size_t workSpaceLimit)
{
  cudnnConvolutionBwdFilterSpecificAlgo = false ;
  cudnnConvolutionBwdFilterPreference = x ;
  cudnnConvolutionBwdFilterWorkSpaceLimit = workSpaceLimit ;
}

void
vl::CudaHelper::setCudnnConvolutionBwdFilterAlgo(hipdnnConvolutionBwdFilterAlgo_t x)

{
  cudnnConvolutionBwdFilterSpecificAlgo = true ;
  cudnnConvolutionBwdFilterAlgo = x ;
}

size_t
vl::CudaHelper::getCudnnConvolutionBwdFilterWorkSpaceUsed() const
{
  return cudnnConvolutionBwdFilterWorkSpaceUsed ;
}

void
vl::CudaHelper::setCudnnConvolutionBwdDataPreference(hipdnnConvolutionBwdDataPreference_t x,
                                                     size_t workSpaceLimit)
{
  cudnnConvolutionBwdDataSpecificAlgo = false ;
  cudnnConvolutionBwdDataPreference = x ;
  cudnnConvolutionBwdDataWorkSpaceLimit = workSpaceLimit ;
}

void
vl::CudaHelper::setCudnnConvolutionBwdDataAlgo(hipdnnConvolutionBwdDataAlgo_t x)
{
  cudnnConvolutionBwdDataSpecificAlgo = true ;
  cudnnConvolutionBwdDataAlgo = x ;
}

size_t
vl::CudaHelper::getCudnnConvolutionBwdDataWorkSpaceUsed() const
{
  return cudnnConvolutionBwdDataWorkSpaceUsed ;
}
#endif

/* -------------------------------------------------------------------
 * CuBLAS Errors
 * ---------------------------------------------------------------- */

static const char *
getCublasErrorMessageFromStatus(hipblasStatus_t status)
{
  switch (status) {
    case HIPBLAS_STATUS_SUCCESS: return "HIPBLAS_STATUS_SUCCESS";
    case HIPBLAS_STATUS_NOT_INITIALIZED: return "HIPBLAS_STATUS_NOT_INITIALIZED";
    case HIPBLAS_STATUS_ALLOC_FAILED: return "HIPBLAS_STATUS_ALLOC_FAILED";
    case HIPBLAS_STATUS_INVALID_VALUE: return "HIPBLAS_STATUS_INVALID_VALUE";
    case HIPBLAS_STATUS_ARCH_MISMATCH: return "HIPBLAS_STATUS_ARCH_MISMATCH";
    case HIPBLAS_STATUS_MAPPING_ERROR: return "HIPBLAS_STATUS_MAPPING_ERROR";
    case HIPBLAS_STATUS_EXECUTION_FAILED: return "HIPBLAS_STATUS_EXECUTION_FAILED";
    case HIPBLAS_STATUS_INTERNAL_ERROR: return "HIPBLAS_STATUS_INTERNAL_ERROR";
#if CUDA_VERSION >= 6000
    case HIPBLAS_STATUS_NOT_SUPPORTED: return "HIPBLAS_STATUS_NOT_SUPPORTED";
#endif
#if CUDA_VERSION >= 6050
    case HIPBLAS_STATUS_UNKNOWN: return "HIPBLAS_STATUS_UNKNOWN";
#endif
  }
  return "CuBLAS unknown status" ;
}

vl::ErrorCode
vl::CudaHelper::catchCublasError(hipblasStatus_t status, char const * description)
{
  /* if there is no CuBLAS error, do not do anything */
  if (status == HIPBLAS_STATUS_SUCCESS) { return vl::VLE_Success ; }

  /* if there is a CuBLAS error, store it */
  lastCublasError = status ;
  std::string message = getCublasErrorMessageFromStatus(status) ;
  if (description) {
    message = std::string(description) + " (" + message + ")" ;
  }
  lastCublasErrorMessage = message ;
  return vl::VLE_Cublas ;
}

hipblasStatus_t
vl::CudaHelper::getLastCublasError() const
{
  return lastCublasError;
}

std::string const&
vl::CudaHelper::getLastCublasErrorMessage() const
{
  return lastCublasErrorMessage ;
}

/* -------------------------------------------------------------------
 * CuDNN Errors
 * ---------------------------------------------------------------- */

#if ENABLE_CUDNN
vl::ErrorCode
vl::CudaHelper::catchCudnnError(hipdnnStatus_t status, char const* description)
{
  /* if there is no CuDNN error, do not do anything */
  if (status == HIPDNN_STATUS_SUCCESS) { return vl::VLE_Success ; }

  /* if there is a CuDNN error, store it */
  lastCudnnError = status ;
  std::string message = hipdnnGetErrorString(status) ;
  if (description) {
    message = std::string(description) + " (" + message + ")" ;
  }
  lastCudnnErrorMessage = message ;
  return vl::VLE_Cudnn ;
}

hipdnnStatus_t
vl::CudaHelper::getLastCudnnError() const
{
  return lastCudnnError;
}

std::string const&
vl::CudaHelper::getLastCudnnErrorMessage() const
{
  return lastCudnnErrorMessage ;
}
#endif

/* -------------------------------------------------------------------
 * Cuda Errors
 * ---------------------------------------------------------------- */

vl::ErrorCode
vl::CudaHelper::catchCudaError(char const* description)
{
  /* if there is no Cuda error, do not do anything */
  hipError_t error = hipPeekAtLastError() ;
  if (error == hipSuccess) { return vl::VLE_Success ; }

  /* if there is a Cuda error, eat it and store it */
  lastCudaError = hipGetLastError() ;
  std::string message = hipGetErrorString(lastCudaError) ;
  if (description) {
    message = std::string(description) + ": " + message ;
  }
  lastCudaErrorMessage = message ;
  return vl::VLE_Cuda ;
}

hipError_t
vl::CudaHelper::getLastCudaError() const
{
  return lastCudaError ;
}

std::string const&
vl::CudaHelper::getLastCudaErrorMessage() const
{
  return lastCudaErrorMessage ;
}



