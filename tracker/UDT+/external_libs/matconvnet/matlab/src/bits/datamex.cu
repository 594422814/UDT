// @file datamex.cu
// @brief Basic data structures (MEX support)
// @author Andrea Vedaldi

/*
Copyright (C) 2015-16 Andrea Vedaldi.
All rights reserved.

This file is part of the VLFeat library and is made available under
the terms of the BSD license (see the COPYING file).
*/

#include "datamex.hpp"
#if ENABLE_GPU
#include "datacu.hpp"
#endif

#include <assert.h>
#ifndef NDEBUG
#include<iostream>
#endif

#include "impl/copy.hpp"

using namespace vl ;
using namespace vl::impl ;

/* ---------------------------------------------------------------- */
/*                                                       MexContext */
/* ---------------------------------------------------------------- */

vl::MexContext::MexContext()
: Context()
#if ENABLE_GPU
, gpuIsInitialized(false)
, canary(NULL)
#endif
{ }

vl::MexContext::~MexContext()
{
#if ENABLE_GPU
  // so that ~Context does not crash if MATLAB resets the GPU in the mean time
  validateGpu() ;
#endif
}

/* ---------------------------------------------------------------- */
/*                                                   GPU management */
/* ---------------------------------------------------------------- */

#if ENABLE_GPU

// Do noting if the GPU is not initialized, otherwise invalidate it
// if needed
vl::ErrorCode
MexContext::validateGpu()
{
  if (!gpuIsInitialized) { return vl::VLE_Success ; }
  gpuIsInitialized = mxGPUIsValidGPUData(canary) ;
  if (!gpuIsInitialized) {
#ifndef NDEBUG
    std::cout<<"MexContext:: GPU reset detected; invalidating the GPU state"<<std::endl ;
#endif
    mxDestroyArray(canary) ;
    canary = NULL ;
    Context::invalidateGpu() ;
  }
  return vl::VLE_Success ;
}

// Initialize GPU; also make sure that it was not reset by MATLAB
vl::ErrorCode
vl::MexContext::initGpu()
{
  validateGpu() ;
  if (!gpuIsInitialized) {
    mwSize dims = 1 ;
    mxInitGPU() ;
    // todo: can mxGPUCreateGPUArray return NULL ?
    mxGPUArray * gpuArray =
    mxGPUCreateGPUArray(1,&dims,mxINT8_CLASS,mxREAL,MX_GPU_DO_NOT_INITIALIZE) ;
    canary = mxGPUCreateMxArrayOnGPU(gpuArray) ;
    mexMakeArrayPersistent(canary) ;
    mxGPUDestroyGPUArray(gpuArray) ;
    gpuIsInitialized = true ;
  }
  return vl::VLE_Success ;
}
#endif

/* ---------------------------------------------------------------- */
/*                                                        MexTensor */
/* ---------------------------------------------------------------- */

/*
 The MexTensor class helps handling MATLAB CPU and GPU arrays.

 The design is somewhat awkward to match MATLAB assumptions.

 The class can either:

 - wrap an existing mxArray (or mxArray + mxGPUArray)
 - or create a new mxArray (or mxArray + mxGPUArray)

 In the second case, the array is released when the destructor is
 called. However, this would normally interfere with MATLAB
 automatic garbage collection upon raising an exception (which
 can happen using mexErrMsgTxt() or, implicitly, when an array
 creation function cannot complete, for example due to a memory error).

 Therefore the constructors make the allocated memory persistent. C++
 guarantees that the arrays are freed upon error in the destructors.

 Note that, upon creating an array, errors such as running out of
 CPU/GPU memory can occur. In this case, MATLAB throws an error
 and quits the MEX file (either implicitly or because we call
 mexErrMsgTxt()). Hence constructors always complete with a well
 defined object.

 */

/* ---------------------------------------------------------------- */
/* Constructing, clearing, destroying                               */
/* ---------------------------------------------------------------- */

vl::MexTensor::MexTensor(MexContext & context)
: context(context),
Tensor(),
array(NULL),
isArrayOwner(false)
#if ENABLE_GPU
, gpuArray(NULL)
#endif
{ }

mxArray *
vl::MexTensor::relinquish()
{
  if (isArrayOwner) {
    isArrayOwner = false ;
    return (mxArray*) array ;
  } else {
    // this is because we may be encapsulating an input argument
    // and we may be trying to return it
    // we should probably use the undocumented
    // extern mxArray *mxCreateSharedDataCopy(const mxArray *pr);
    return mxDuplicateArray(array) ;
  }
}

void
vl::MexTensor::makePersistent()
{
  assert(isArrayOwner) ;
  mexMakeArrayPersistent((mxArray*)array) ;
}

void
vl::MexTensor::clear()
{
#if ENABLE_GPU
  if (gpuArray) {
    mxGPUDestroyGPUArray(gpuArray) ;
    gpuArray = NULL ;
  }
#endif
  if (isArrayOwner) {
    if (array) {
      mxDestroyArray((mxArray*)array) ;
      array = NULL ;
    }
    isArrayOwner = false ;
  }
  memory = NULL ;
  memorySize = 0 ;
  deviceType = vl::VLDT_CPU ;
  vl::TensorShape::clear() ;
}

vl::MexTensor::~MexTensor()
{
  clear() ;
}

/* ---------------------------------------------------------------- */
/* init with optional zero filling                                  */
/* ---------------------------------------------------------------- */

vl::ErrorCode
vl::MexTensor::initHelper(DeviceType newDeviceType, DataType newDataType,
                          TensorShape const & newShape, bool fillWithZeros)
{
  clear() ;

  // assign dimensions
  mwSize dimensions [VL_TENSOR_SHAPE_MAX_NUM_DIMENSIONS] ;
  for (int k = 0 ; k < newShape.getNumDimensions() ; ++k) {
    dimensions[k] = (mwSize)newShape.getDimension(k) ;
  }

  // compute the size in bytes
  mwSize newMemorySize = newShape.getNumElements() ;
  mxClassID classID ;
  switch (newDataType) {
    case VLDT_Float:
      newMemorySize *= sizeof(DataTypeTraits<VLDT_Float>::type) ;
      classID = mxSINGLE_CLASS ;
      break ;
#ifdef ENABLE_DOUBLE
    case VLDT_Double:
      newMemorySize *= sizeof(DataTypeTraits<VLDT_Double>::type) ;
      classID = mxDOUBLE_CLASS ;
      break ;
#endif
    default:
      abort() ;
  }

  // allocate the memory on CPU or GPU
  void * newMemory = NULL ;
  mxArray * newArray = NULL ;
#if ENABLE_GPU
  mxGPUArray* newGpuArray = NULL ;
#endif

  if (newDeviceType == vl::VLDT_CPU) {
    if (fillWithZeros) {
      newArray = mxCreateNumericArray(4, dimensions, classID, mxREAL) ;
      newMemory = mxGetData(newArray) ;
    } else {
      mwSize dimensions_ [1] = {0} ;
      newMemory = mxMalloc(newMemorySize) ;
      newArray = mxCreateNumericArray(1, dimensions_,
                                      classID,
                                      mxREAL) ;
      mxSetData(newArray, newMemory) ;
      mxSetDimensions(newArray, dimensions, newShape.getNumDimensions()) ;
    }
  }
#ifdef ENABLE_GPU
  else {
    context.initGpu() ;
    newGpuArray = mxGPUCreateGPUArray(newShape.getNumDimensions(), dimensions,
                                      classID,
                                      mxREAL,
                                      fillWithZeros ? MX_GPU_INITIALIZE_VALUES : MX_GPU_DO_NOT_INITIALIZE) ;
    newArray = mxGPUCreateMxArrayOnGPU(newGpuArray) ;
    newMemory = mxGPUGetData(newGpuArray) ;
  }
#else
  else {
    abort() ;
  }
#endif

  //mexMakeArrayPersistent(newArray) ; // avoid double free with MATALB garbage collector upon error
  TensorShape::operator=(newShape) ;
  deviceType = newDeviceType ;
  dataType = newDataType ;
  memory = newMemory ;
  memorySize = newMemorySize ;
  array = newArray ;
  isArrayOwner = true ;
#if ENABLE_GPU
  gpuArray = newGpuArray ;
#endif
  return vl::VLE_Success ;
}

/* ---------------------------------------------------------------- */
/*                                                          init*() */
/* ---------------------------------------------------------------- */

vl::ErrorCode
vl::MexTensor::init(DeviceType newDeviceType,
                    DataType newDataType,
                    TensorShape const & newShape)
{
  return initHelper(newDeviceType, newDataType, newShape, false) ;
}

vl::ErrorCode
vl::MexTensor::initWithZeros(DeviceType newDeviceType,
                             DataType newDataType,
                             TensorShape const & newShape)
{
  return initHelper(newDeviceType, newDataType, newShape, true) ;
}

vl::ErrorCode
vl::MexTensor::initWithValue(DeviceType newDeviceType,
                             DataType newDataType,
                             TensorShape const & newShape,
                             double value)
{
  if (value == 0) {
    return initHelper(newDeviceType, newDataType, newShape, true) ;
  } else {
    vl::ErrorCode error = initHelper(newDeviceType, newDataType, newShape, false) ;
    if (error != VLE_Success) { return error ; }
    size_t const n = getNumElements() ;
    if (newDeviceType == vl::VLDT_CPU) {
      switch (newDataType) {
        case VLDT_Float: error = operations<vl::VLDT_CPU,float>::fill((float*)memory, n, (float)value) ; break ;
#ifdef ENABLE_DOUBLE
        case VLDT_Double: error = operations<vl::VLDT_CPU,double>::fill((double*)memory, n, (double)value) ; break ;
#endif
        default: abort() ;
      }
    }
#ifdef ENABLE_GPU
    else {
      switch (newDataType) {
        case VLDT_Float: error = operations<vl::VLDT_GPU,float>::fill((float*)memory, n, (float)value) ; break ;
#ifdef ENABLE_DOUBLE
        case VLDT_Double: error = operations<vl::VLDT_GPU,double>::fill((double*)memory, n, (double)value) ; break ;
#endif
        default: abort() ;
      }
      if (error == VLE_Cuda) {
        hipError_t error = hipGetLastError() ;
        clear() ;
        mexErrMsgTxt((std::string("MexTensor: fill [CUDA error: ")
                      + hipGetErrorString(error)
                      + "]"
                      ).c_str()) ;
      }
    }
#endif
  }
  return vl::VLE_Success ;
}

/* ---------------------------------------------------------------- */
/* init by wrapping a given array                                   */
/* ---------------------------------------------------------------- */

vl::ErrorCode
vl::MexTensor::init(mxArray const * array_)
{
  clear() ;
  if (array_ == NULL) { return vl::VLE_Success ; } // empty

  vl::DeviceType newDeviceType ;
  vl::DataType newDataType ;
  void const * newMemory = NULL ;
  mxArray * newArray = (mxArray*)array_ ;
#if ENABLE_GPU
  mxGPUArray* newGpuArray = NULL ;
#endif

  mwSize const * newDimensions ;
  mwSize newNumDimensions ;
  mxClassID newClassID ;

#ifdef ENABLE_GPU
  context.initGpu() ;
  if (mxIsGPUArray(array_)) {
    newDeviceType = vl::VLDT_GPU ;
    newGpuArray = (mxGPUArray*) mxGPUCreateFromMxArray(newArray) ;
    newMemory = mxGPUGetDataReadOnly(newGpuArray) ;
    newClassID = mxGPUGetClassID(newGpuArray) ;
    newDimensions = mxGPUGetDimensions(newGpuArray) ;
    newNumDimensions = mxGPUGetNumberOfDimensions(newGpuArray) ;
  } else
#endif
  {
    if (!mxIsNumeric(newArray)) {
      mexErrMsgTxt("An input is not a numeric array (or GPU support not compiled).") ;
    }
    newDeviceType = VLDT_CPU ;
    newMemory = mxGetData(newArray) ;
    newClassID = mxGetClassID(newArray) ;
    newDimensions = mxGetDimensions(newArray) ;
    newNumDimensions = mxGetNumberOfDimensions(newArray) ;
  }

  if (newNumDimensions >= VL_TENSOR_SHAPE_MAX_NUM_DIMENSIONS) {
#if ENABLE_GPU
    if (newGpuArray) {
      mxGPUDestroyGPUArray(newGpuArray) ;
      newGpuArray = NULL ;
    }
#endif
    mexErrMsgTxt("An input has more than the maximum number of allowed dimensions.") ;
  }

  numDimensions = newNumDimensions ;
  for (int k = 0 ; k < numDimensions ; ++k) {
    setDimension(k, newDimensions[k]) ;
  }

  size_t newMemorySize = getNumElements() ;

  switch (newClassID) {
    case mxSINGLE_CLASS:
      newDataType = VLDT_Float ;
      newMemorySize *= sizeof(DataTypeTraits<VLDT_Float>::type) ;
      break ;

#ifdef ENABLE_DOUBLE
    case mxDOUBLE_CLASS:
      newDataType = VLDT_Double ;
      newMemorySize *= sizeof(DataTypeTraits<VLDT_Double>::type) ;
      break ;
#endif

    default:
      if (isEmpty()) {
        newDataType = VLDT_Float ;
        newMemorySize = 0 ;
        break ;
      }
#ifdef ENABLE_DOUBLE
      mexErrMsgTxt("An input is neither SINGLE or DOUBLE nor it is empty.") ;
#else
      mexErrMsgTxt("An input is neither SINGLE nor empty.") ;
#endif
      break ;
  }

  deviceType = newDeviceType ;
  dataType = newDataType ;
  memory = (void*)newMemory ;
  memorySize = newMemorySize ;
  array = newArray ;
  isArrayOwner = false ;
#if ENABLE_GPU
  gpuArray = newGpuArray ;
#endif

  return vl::VLE_Success ;
}

size_t
vl::MexTensor::getMemorySize() const
{
  return memorySize ;
}

void vl::print(char const * str, vl::MexTensor const & tensor)
{
  size_t size = tensor.getMemorySize() ;
  double scaled ;
  size_t const * dimensions = tensor.getDimensions() ;
  const char * units ;
  const char * type ;
  if (size < 1024) {
    scaled = size ;
    units = "B" ;
  } else if (size < 1024*1024) {
    scaled = size / 1024.0 ;
    units = "KB" ;
  } else if (size < 1024*1024*1024) {
    scaled = size / (1024.0 * 1024.0) ;
    units = "MB" ;
  } else {
    scaled = size / (1024.0 * 1024.0 * 1024.0) ;
    units = "GB" ;
  }
  const char * dev = "" ;
  switch (tensor.getDeviceType()) {
    case vl::VLDT_CPU : dev = "CPU" ; break ;
    case vl::VLDT_GPU : dev = "GPU" ; break ;
  }
  switch (tensor.getDataType()) {
    case vl::VLDT_Float: type = "float" ; break ;
    case vl::VLDT_Double: type = "double" ; break ;
    case vl::VLDT_Char: type = "char" ; break ;
    default: type = "uknown type" ;
  }
  mexPrintf("%s[", str) ;
  for (int k = 0 ; k < tensor.getNumDimensions() ; ++k) {
    mexPrintf("%d ", dimensions[k]) ;
  }
  mexPrintf("| %s %.1f%s %s]\n",
            type,
            scaled,
            units,
            dev);
}
