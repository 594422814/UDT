#include "hip/hip_runtime.h"
// @file copy_gpu.cu
// @brief Copy and other data operations (GPU)
// @author Andrea Vedaldi

/*
Copyright (C) 2015-16 Andrea Vedaldi.
All rights reserved.

This file is part of the VLFeat library and is made available under
the terms of the BSD license (see the COPYING file).
*/

#include "copy.hpp"
#include "../datacu.hpp"
#include <string.h>

template<typename type> __global__ void
fill_kernel (type * data, type value, size_t size)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x ;
  if (index < size) data[index] = value ;
}

namespace vl { namespace impl {

  template <typename type>
  struct operations<vl::VLDT_GPU, type>
  {
    typedef type data_type ;

    static vl::ErrorCode
    copy(data_type * dest,
         data_type const * src,
         size_t numElements)
    {
      hipMemcpy(dest, src, numElements * sizeof(data_type), hipMemcpyDeviceToDevice) ;
      return VLE_Success ;
    }

    static vl::ErrorCode
    fill(data_type * dest,
         size_t numElements,
         data_type value)
    {
      fill_kernel <data_type>
      <<<divideAndRoundUp(numElements, (size_t)VL_CUDA_NUM_THREADS), VL_CUDA_NUM_THREADS>>>
      (dest, numElements, value) ;

      hipError_t error = hipGetLastError() ;
      if (error != hipSuccess) {
        return VLE_Cuda ;
      }
      return VLE_Success ;
    }
  } ;

} }

template struct vl::impl::operations<vl::VLDT_GPU, float> ;

#ifdef ENABLE_DOUBLE
template struct vl::impl::operations<vl::VLDT_GPU, double> ;
#endif