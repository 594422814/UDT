/** @file vl_cudatool.cu
 ** @brief Low-level CUDA tricks.
 ** @author Andrea Vedaldi
 **/

/*
Copyright (C) 2016 Andrea Vedaldi.
All rights reserved.

This file is part of the VLFeat library and is made available under
the terms of the BSD license (see the COPYING file).
*/

#include "bits/mexutils.h"

#include "gpu/mxGPUArray.h"
#include "bits/mexutils.h"
#include "hip/hip_runtime_api.h"

enum {
  IN_COMMAND, IN_ARG1, IN_ARG2, IN_END
} ;

enum {
  OUT_END
} ;

typedef struct Memory_ {
  void * address ;
  size_t size ;
  mxClassID classID ;
} Memory ;

Memory
getMemoryFromArray(const mxArray * array)
{
  Memory mem ;
  if (mxIsGPUArray(array)) {
    mxGPUArray* garray = (mxGPUArray*) mxGPUCreateFromMxArray(array) ;
    mem.address = (void*) mxGPUGetDataReadOnly(garray) ;
    mem.size = mxGPUGetNumberOfElements(garray) ;
    mem.classID = mxGPUGetClassID(garray) ;
    mxGPUDestroyGPUArray(garray) ;
  } else {
    mem.address = mxGetData((mxArray*)array) ;
    mem.size = mxGetNumberOfElements(array) ;
    mem.classID = mxGetClassID(array) ;
  }

  switch (mem.classID) {
  case mxDOUBLE_CLASS: mem.size *= sizeof(double) ; break ;
  case mxSINGLE_CLASS: mem.size *= sizeof(float) ; break ;
  default:
    vlmxError(VLMXE_IllegalArgument, "Data type unsupported.") ;
  }
  return mem ;
}

void mexFunction(int nout, mxArray *out[],
                 int nin, mxArray const *in[])
{

  if (nin < 1) {
    vlmxError(VLMXE_IllegalArgument, "Not enough arguments.") ;
  }

  if (!vlmxIsString(in[0], -1)) {
    vlmxError(VLMXE_IllegalArgument, "COMMAND is not a string.") ;
  }

  if (vlmxCompareToStringI(in[0], "getMemory") == 0) {
    if (nin != 2) {
      vlmxError(VLMXE_IllegalArgument, "Incorrect number of arguments for 'getMemory'.") ;
    }
    Memory mem = getMemoryFromArray(in[1]) ;
    out[0] = mxCreateNumericMatrix(1,2,mxUINT64_CLASS,mxREAL) ;
    size_t* x = (size_t*)mxGetPr(out[0]) ;
    x[0] = (size_t)mem.address ;
    x[1] = mem.size ;
  }
  else if (vlmxCompareToStringI(in[0], "cudaRegister") == 0) {
    if (nin != 2) {
      vlmxError(VLMXE_IllegalArgument, "Incorrect number of arguments for 'cudaRegister'.") ;
    }
    Memory mem = getMemoryFromArray(in[1]) ;
    hipError_t err = hipHostRegister(mem.address,
                                       mem.size,
                                       hipHostRegisterDefault);
    if (err != hipSuccess) {
      vlmxWarning(VLMXE_Execution, "hipHostRegister failied\n");
    }
  }
  else if (vlmxCompareToStringI(in[0], "cudaUnregister") == 0) {
    if (nin != 2) {
      vlmxError(VLMXE_IllegalArgument, "Incorrect number of arguments for 'cudaUnregister'.") ;
    }
    Memory mem = getMemoryFromArray(in[1]) ;
    hipError_t err = hipHostUnregister(mem.address) ;
    if (err != hipSuccess) {
      mexWarnMsgTxt("hipHostUnregister failied\n");
    }
  }
  else if (vlmxCompareToStringI(in[0], "cudaCopyDeviceToHost") == 0) {
    if (nin != 3) {
      vlmxError(VLMXE_IllegalArgument, "Incorrect number of arguments for 'cudaCopyDeviceToHost'") ;
    }
    Memory mem = getMemoryFromArray(in[1]) ;
    Memory gmem = getMemoryFromArray(in[2]) ;
    hipMemcpy(mem.address,
               gmem.address,
               gmem.size,
               hipMemcpyDeviceToHost);
  }
  else if (vlmxCompareToStringI(in[0], "cudaCopyDeviceToHostAsync") == 0) {
    if (nin != 3) {
      vlmxError(VLMXE_IllegalArgument, "Incorrect number of arguments for 'cudaCopyDeviceToHostAsync'.") ;
    }
    Memory mem = getMemoryFromArray(in[1]) ;
    Memory gmem = getMemoryFromArray(in[2]) ;
    hipMemcpyAsync(mem.address,
                    gmem.address,
                    gmem.size,
                    hipMemcpyDeviceToHost,
                    0);
  }
  else if (vlmxCompareToStringI(in[0], "hipProfilerStart") == 0) {
    hipProfilerStart() ;
    mexPrintf("Enabled CUDA profiler.\n") ;
  }
  else if (vlmxCompareToStringI(in[0], "hipProfilerStop") == 0) {
    hipProfilerStop() ;
    mexPrintf("Disabled CUDA profiler.\n") ;
  }
  else {
    vlmxError(VLMXE_IllegalArgument, "Unrecognized command COMMAND.") ;
  }
}
